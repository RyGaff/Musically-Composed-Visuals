#include "hip/hip_runtime.h"
/**
 * CS470 - Final Project - Audio Visuals Team
 *
 * Audio Processor Program
 *
 * Team: Matthew Dim, Ryan Gaffney, Ian Lips, Justin Choi
 */
#include <cstdint>
#include <iostream>
#include <vector>
#include <complex>
#include <cmath>
#include <fstream>
//#include <bits/stdc++.h>
#include <string>

#include <hip/hip_complex.h>
//typedef complex<double> cd;

#include "timer.h"
using namespace std;

// Not sure if this needs to be used yet
struct complexD{
    double real;
    double imag;

    __device__ complexD(double r, double i): real(r), imag(i){}
    __device__ complexD operator*(complexD& a){
        return complexD(real * a.real - imag * a.imag, imag * a.real + real * a.imag);
    }
    __device__ complexD operator+(complexD& a){
        return complexD(real + a.real, imag + a.imag);
    }
};

//typedef complexD cd;

struct WAV_HEADER{
    // Riff
    uint8_t chunkID[4];
    uint32_t chunkSize;
    uint8_t format[4];
    // FMT
    uint8_t subchunk1ID[4];
    uint32_t subchunk1Size;
    uint16_t audioFormat;
    uint16_t numChannels;
    uint32_t samepleRate;
    uint32_t byteRate;
    uint16_t blockAlign;
    uint16_t bitsPerSample;
    // Data
    uint8_t subchunk2ID[4];
    uint32_t subchunk2Size;
};

int thread_count;

vector<complex<double>> convertWavDataToComplexVector(vector<int16_t> d){
    vector<complex<double>> output;
    for(int16_t i : d){
        output.push_back(complex<double>(i,0));
    }
    return output;
}

vector<complex<double>> getDataFromWav(const std::string &file_path){
    ifstream wav(file_path);
    struct WAV_HEADER whr;

    if(!wav.is_open()){
        throw "File Could Not be Opened!";
    }
    
    // Riff
    wav.read((char*)&whr.chunkID, sizeof(whr.chunkID)); 
    wav.read((char*)&whr.chunkSize, sizeof(whr.chunkSize));
    wav.read((char*)&whr.format, sizeof(whr.format));

    // fmt
    wav.read((char*)&whr.subchunk1ID, sizeof(whr.subchunk1ID));
    wav.read((char*)&whr.subchunk1Size, sizeof(whr.subchunk1Size));
    wav.read((char*)&whr.audioFormat,sizeof(whr.audioFormat));
    wav.read((char*)&whr.numChannels,sizeof(whr.numChannels));
    wav.read((char*)&whr.samepleRate,sizeof(whr.samepleRate));
    wav.read((char*)&whr.byteRate,sizeof(whr.byteRate));
    wav.read((char*)&whr.blockAlign,sizeof(whr.blockAlign));
    wav.read((char*)&whr.bitsPerSample,sizeof(whr.bitsPerSample));

    // Data
    wav.read((char*)&whr.subchunk2ID,sizeof(whr.subchunk2ID));
    wav.read((char*)&whr.subchunk2Size,sizeof(whr.subchunk2Size));

    vector<int16_t> data(whr.subchunk2Size);
    
    for(uint32_t i = 0; i < whr.subchunk2Size; i++){
        wav.read((char*)&data[i],sizeof(data[i]));
    }

    // Read audio data
    wav.close();
    
    return convertWavDataToComplexVector(data);
}

vector<hipFloatComplex> convertWavDataToComplexVector2(vector<int16_t> d){
    vector<hipFloatComplex> output;
    for(int16_t i : d){
        output.push_back(make_hipFloatComplex(i,0));
    }
    return output;
}

vector<hipFloatComplex> getDataFromWav2(const std::string &file_path){
    ifstream wav(file_path);
    struct WAV_HEADER whr;

    if(!wav.is_open()){
        throw "File Could Not be Opened!";
    }
    
    // Riff
    wav.read((char*)&whr.chunkID, sizeof(whr.chunkID)); 
    wav.read((char*)&whr.chunkSize, sizeof(whr.chunkSize));
    wav.read((char*)&whr.format, sizeof(whr.format));

    // fmt
    wav.read((char*)&whr.subchunk1ID, sizeof(whr.subchunk1ID));
    wav.read((char*)&whr.subchunk1Size, sizeof(whr.subchunk1Size));
    wav.read((char*)&whr.audioFormat,sizeof(whr.audioFormat));
    wav.read((char*)&whr.numChannels,sizeof(whr.numChannels));
    wav.read((char*)&whr.samepleRate,sizeof(whr.samepleRate));
    wav.read((char*)&whr.byteRate,sizeof(whr.byteRate));
    wav.read((char*)&whr.blockAlign,sizeof(whr.blockAlign));
    wav.read((char*)&whr.bitsPerSample,sizeof(whr.bitsPerSample));

    // Data
    wav.read((char*)&whr.subchunk2ID,sizeof(whr.subchunk2ID));
    wav.read((char*)&whr.subchunk2Size,sizeof(whr.subchunk2Size));

    vector<int16_t> data(whr.subchunk2Size);
    
    for(uint32_t i = 0; i < whr.subchunk2Size; i++){
        wav.read((char*)&data[i],sizeof(data[i]));
    }

    // Read audio data
    wav.close();
    
    return convertWavDataToComplexVector2(data);
}

void dft(vector<complex<double>> signal,vector<complex<double>>& output){
    for(uint64_t k = 0; k < signal.size(); k++){
        complex<double> ans(0,0);
        for(uint64_t t = 0; t < signal.size(); t++){
            double angle = (-2 * M_PI * t * k) / signal.size(); 	
            ans += signal[t] * exp(complex<double>(0,angle));
        }
        output.push_back(ans);
    }
}


/*
 * Power of 2 helper function.
 */
constexpr int findNextPowerOfTwo(int N){
    N--;
    N |= N >> 1;
    N |= N >> 2;
    N |= N >> 4;
    N |= N >> 8;
    N |= N >> 16;
    N++;
    return N;
}

/*
 * Power of 2 helper function.
 */
constexpr bool isPowerOfTwo(int N){
    return (N & (N-1)) == 0;
}

/*
 * Ensure that signal length is a power of two
 */
void transformSignal(vector<complex<double>>& signal){
    int diff = isPowerOfTwo(signal.size()) ? 0 : findNextPowerOfTwo(signal.size()) - signal.size();
    if(diff == 0) {
        return;
    }else{
        for(int i = 0; i < diff; i++){
            signal.push_back(0);	
        }
    }
}

/*
 * Ensure that signal length is a power of two
 */
void transformSignal(vector<hipFloatComplex>& signal){
    int diff = isPowerOfTwo(signal.size()) ? 0 : findNextPowerOfTwo(signal.size()) - signal.size();
    if(diff == 0) {
        return;
    }else{
        for(int i = 0; i < diff; i++){
            signal.push_back(make_hipFloatComplex(0,0));	
        }
    }
}


/*
 * Perform a Fast Fourier Transform (FFT) on audio data.
 *
 * Output will be a vector.
 */
void fft(vector<complex<double>>& signal){
// #  pragma omp parallel default(none) shared(signal)
    {
   
        // Thread count
#       ifdef _OPENMP
        thread_count = omp_get_thread_num();
        printf("Thread Count Set At: %d", thread_count);
#       else
        thread_count = 1;
#       endif

        transformSignal(signal);
        int N = signal.size();

        if(N == 1) return; // Base case

        vector<complex<double>> even(N/2), odd(N/2); 
// #       pragma omp for
        for(int i = 0; 2 * i < N; i++){
            even[i] = signal[2*i];
            odd[i] = signal[2*i+1];
        }

        fft(even);
        fft(odd);

        double angle = 2 * M_PI / N;
        complex<double> w_n(cos(angle),sin(angle));
        complex<double> w(1);
//#       pragma omp for shared(even, odd) reduction(*:w)
        for(int i = 0; 2 * i < N; i++){
             signal[i] = even[i] + w * odd[i];
             signal[i + N/2] = even[i] - w * odd[i];
             w *= w_n;
        } 
    }
}

// https://en.wikipedia.org/wiki/Cooley%E2%80%93Tukey_FFT_algorithm
// cd is a complex double
/*
 * Bit reversal algorithm for the iterative version of fft.
 * this is needed because we are doing a bottom up implementation isntead of 
 * top down like we did with the recursive fft
 *  
 */
__device__ unsigned int bit_reversal(unsigned int i, int log2n){
    int rev = 0;
    for (int j = 0; j < log2n; j++) {
        rev <<= 1;
        rev |= (i & 1);
        i >>= 1;
    }
    return rev;
}
/*
 * My brain hurts
 */
__global__ void iterative_fft_kernel(const hipFloatComplex* a, hipFloatComplex* A, int log2n){
    
    uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    A[i] = a[bit_reversal(i, log2n)];

    __syncthreads();

    for (int s = 1; s <= log2n; ++s) {
        int m2 = 1 << (s - 1); // m2 = m/2 -1#    

        int j = threadIdx.x % m2;
        int k = threadIdx.x / m2 * (1 << s);

        hipFloatComplex u = A[k+j];
        float tempr;
        float tempi;

        sincosf(-3.1415926536 * j, &tempi, &tempr);
        hipFloatComplex w = make_hipFloatComplex(tempr, tempi);
        hipFloatComplex t = hipCmulf(w, A[k+j + m2]);
        A[k+j] = hipCaddf(u,t);
        A[k+j + m2] = hipCsubf(u,t);
        __syncthreads();
    }
}
/*
 * Handles the cuda operations and calls iterative_fft_kernel
 */
int fft_cuda(const hipFloatComplex* a, hipFloatComplex* A, int log2n, int N){    
    // Allocate memory on the cuda device
    hipFloatComplex* a0;
    hipFloatComplex* A0;
    hipMalloc((void **)&a0, sizeof(hipFloatComplex) * N);
    hipMalloc((void **)&A0, sizeof(hipFloatComplex) * N);
    
    hipMemcpy(a0, a, sizeof(hipFloatComplex) * N, hipMemcpyHostToDevice);

    //Just putting this here for now dont feel like making testsing stuff atm
    //Just gets the max amount of threads possible given the cuda device
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    int size = N >> 1;
    int block_size = min(size, properties.maxThreadsPerBlock);
    dim3 block(block_size, 1);
    dim3 grid(16,1);

    iterative_fft_kernel<<<grid, block>>>(a0, A0, log2n);
    hipMemcpy(A, A0, sizeof(hipFloatComplex)* N, hipMemcpyDeviceToHost);
    
    hipFree(a0);
    hipFree(A0);

    return 0;
}

/*
 * Python helper function to plot fourier transform
 */
void plotOutputData(){
    system("python3 ./python-stuffs/plotter.py");
}

// void normalizeCSVFile(const vector<cd>& out, double max_real, double max_imag, const string fileName = "coords.csv"){
//     ofstream outFile("normalized_" + fileName);
//     for (complex<double> i : out){
//         outFile << i.real()/max_real << "," << i.imag()/max_imag << "\n";
//     }
//     outFile.close();

// }
/*
 * Write data to a CSV file
 *
 * File will be parsed in visualizer
 * 
 * Return: The number of complex numbers ie number of lines, will make wrapping with the visalizer easier
 */
int writeDataToCSVFile(const vector<complex<double>>& out, const string fileName = "coords.csv"){

    ofstream outFile(fileName);
    outFile << "x,y" << "\n";
    int count = 0;
//    double max_real = numeric_limits<double>::min();
//    double max_imag = numeric_limits<double>::min();
    for(complex<double> i : out){
        count++;
        outFile << i.real() << "," << i.imag() << "\n";

//        if (i.real() > max_real){
//            max_real = i.real();
//        } 
//        if (i.imag() > max_imag){
//            max_imag = i.imag();
//        }
    }

    outFile.close();

    // plotOutputData();
    // normalizeCSVFile(out, max_real, max_imag, fileName);
    return count;
}

/*
 * Write data to a CSV file
 *
 * File will be parsed in visualizer
 * 
 * Return: The number of complex numbers ie number of lines, will make wrapping with the visalizer easier
 */
int writeDataToCSVFile(hipFloatComplex* out, int outsize,const string fileName = "coords.csv"){

    ofstream outFile(fileName);
    outFile << "x,y" << "\n";
    int count = 0;
  //  double max_real = numeric_limits<double>::min();
   // double max_imag = numeric_limits<double>::min();
    for(int i = 0; i < outsize; i++){
        count++;
        outFile << out[i].x << "," << out[i].y << "\n";
    }
  //      if (out[i].x > max_real){
  //         max_real = out[i].x;
  //      } 
  //      if (out[i].y > max_imag){
  //          max_imag = out[i].y;
  //      }
  //  }

    outFile.close();

    // plotOutputData();
    // normalizeCSVFile(out, max_real, max_imag, fileName);
    return count;
}




int main(int argc,const char** argv){

    cin.tie(0);

    // Read input file
    if (argc != 3) {
   	printf("Usage: %s <input-wav-file-name> <output-csv-file-name>\n args = %d", argv[0],argc); 
	exit(EXIT_FAILURE);
    }

    std::string file_name = argv[1];
    std::string csv_name  = argv[2];

   // vector<complex<double>> output = getDataFromWav(file_name);    
    vector<hipFloatComplex> output = getDataFromWav2(file_name);    
    transformSignal(output); // Ensure that output size is a power of 2

    // Convert the vector to array, yes I know not optimal
    hipFloatComplex in[output.size()];
    copy(output.begin(),output.end(), in);

    // Add a timer to test for parallelism
    // Add a barrier if needed
    // vector<complex<double>> iterative_out(output.size());
    hipFloatComplex* out;
    out = (hipFloatComplex*)calloc(output.size(),sizeof(hipFloatComplex));
    
    int log2n = log2(output.size());
    // Init Cuda stuff
    START_TIMER(fft);
    // fft(output);
    // for (unsigned int i = 0; i < output.size(); ++i) {
    //         out[i] = in[bit_reversal(i, log2n)];
    //     }
    fft_cuda(in, out, log2n, output.size());
    STOP_TIMER(fft);
    
    // Serial for now - TODO: Add OMP def
    printf("Thread Count: %d - FFT Type: Misc for now - FFT Time: %lfs\n", thread_count, GET_TIMER(fft));
    writeDataToCSVFile(out, output.size(), csv_name);
    // writeDataToCSVFile(output, "recursive_" + csv_name);

    return EXIT_SUCCESS;
}