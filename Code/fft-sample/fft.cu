#include "hip/hip_runtime.h"
/**
 * CS470 - Final Project - Audio Visuals Team
 *
 * Audio Processor Program
 *
 * Team: Matthew Dim, Ryan Gaffney, Ian Lips, Justin Choi
 */
#include <cstdint>
#include <iostream>
#include <vector>
#include <complex>
#include <cmath>
#include <fstream>
//#include <bits/stdc++.h>
#include <string>

#include <hip/hip_complex.h>
//typedef complex<double> cd;

#include "timer.h"
using namespace std;

// Not sure if this needs to be used yet
struct complexD{
    double real;
    double imag;

    __device__ complexD(double r, double i): real(r), imag(i){}
    __device__ complexD operator*(complexD& a){
        return complexD(real * a.real - imag * a.imag, imag * a.real + real * a.imag);
    }
    __device__ complexD operator+(complexD& a){
        return complexD(real + a.real, imag + a.imag);
    }
};

//typedef complexD cd;

struct WAV_HEADER{
    // Riff
    uint8_t chunkID[4];
    uint32_t chunkSize;
    uint8_t format[4];
    // FMT
    uint8_t subchunk1ID[4];
    uint32_t subchunk1Size;
    uint16_t audioFormat;
    uint16_t numChannels;
    uint32_t samepleRate;
    uint32_t byteRate;
    uint16_t blockAlign;
    uint16_t bitsPerSample;
    // Data
    uint8_t subchunk2ID[4];
    uint32_t subchunk2Size;
};

int thread_count;

vector<complex<double>> convertWavDataToComplexVector(vector<int16_t> d){
    vector<complex<double>> output;
    for(int16_t i : d){
        output.push_back(complex<double>(i,0));
    }
    return output;
}

vector<complex<double>> getDataFromWav(const std::string &file_path){
    ifstream wav(file_path);
    struct WAV_HEADER whr;

    if(!wav.is_open()){
        throw "File Could Not be Opened!";
    }
    
    // Riff
    wav.read((char*)&whr.chunkID, sizeof(whr.chunkID)); 
    wav.read((char*)&whr.chunkSize, sizeof(whr.chunkSize));
    wav.read((char*)&whr.format, sizeof(whr.format));

    // fmt
    wav.read((char*)&whr.subchunk1ID, sizeof(whr.subchunk1ID));
    wav.read((char*)&whr.subchunk1Size, sizeof(whr.subchunk1Size));
    wav.read((char*)&whr.audioFormat,sizeof(whr.audioFormat));
    wav.read((char*)&whr.numChannels,sizeof(whr.numChannels));
    wav.read((char*)&whr.samepleRate,sizeof(whr.samepleRate));
    wav.read((char*)&whr.byteRate,sizeof(whr.byteRate));
    wav.read((char*)&whr.blockAlign,sizeof(whr.blockAlign));
    wav.read((char*)&whr.bitsPerSample,sizeof(whr.bitsPerSample));

    // Data
    wav.read((char*)&whr.subchunk2ID,sizeof(whr.subchunk2ID));
    wav.read((char*)&whr.subchunk2Size,sizeof(whr.subchunk2Size));

    vector<int16_t> data(whr.subchunk2Size);
    
    for(uint32_t i = 0; i < whr.subchunk2Size; i++){
        wav.read((char*)&data[i],sizeof(data[i]));
    }

    // Read audio data
    wav.close();
    
    return convertWavDataToComplexVector(data);
}

vector<hipDoubleComplex> convertWavDataToComplexVector2(vector<int16_t> d){
    vector<hipDoubleComplex> output;
    for(int16_t i : d){
        output.push_back(make_hipDoubleComplex(i,0));
    }
    return output;
}

vector<hipDoubleComplex> getDataFromWav2(const std::string &file_path){
    ifstream wav(file_path);
    struct WAV_HEADER whr;

    if(!wav.is_open()){
        throw "File Could Not be Opened!";
    }
    
    // Riff
    wav.read((char*)&whr.chunkID, sizeof(whr.chunkID)); 
    wav.read((char*)&whr.chunkSize, sizeof(whr.chunkSize));
    wav.read((char*)&whr.format, sizeof(whr.format));

    // fmt
    wav.read((char*)&whr.subchunk1ID, sizeof(whr.subchunk1ID));
    wav.read((char*)&whr.subchunk1Size, sizeof(whr.subchunk1Size));
    wav.read((char*)&whr.audioFormat,sizeof(whr.audioFormat));
    wav.read((char*)&whr.numChannels,sizeof(whr.numChannels));
    wav.read((char*)&whr.samepleRate,sizeof(whr.samepleRate));
    wav.read((char*)&whr.byteRate,sizeof(whr.byteRate));
    wav.read((char*)&whr.blockAlign,sizeof(whr.blockAlign));
    wav.read((char*)&whr.bitsPerSample,sizeof(whr.bitsPerSample));

    // Data
    wav.read((char*)&whr.subchunk2ID,sizeof(whr.subchunk2ID));
    wav.read((char*)&whr.subchunk2Size,sizeof(whr.subchunk2Size));

    vector<int16_t> data(whr.subchunk2Size);
    
    for(uint32_t i = 0; i < whr.subchunk2Size; i++){
        wav.read((char*)&data[i],sizeof(data[i]));
    }

    // Read audio data
    wav.close();
    
    return convertWavDataToComplexVector2(data);
}



/*
 * Power of 2 helper function.
 */
constexpr int findNextPowerOfTwo(int N){
    N--;
    N |= N >> 1;
    N |= N >> 2;
    N |= N >> 4;
    N |= N >> 8;
    N |= N >> 16;
    N++;
    return N;
}

/*
 * Power of 2 helper function.
 */
constexpr bool isPowerOfTwo(int N){
    return (N & (N-1)) == 0;
}

/*
 * Ensure that signal length is a power of two
 */
void transformSignal(vector<complex<double>>& signal){
    int diff = isPowerOfTwo(signal.size()) ? 0 : findNextPowerOfTwo(signal.size()) - signal.size();
    if(diff == 0) {
        return;
    }else{
        for(int i = 0; i < diff; i++){
            signal.push_back(0);	
        }
    }
}

/*
 * Ensure that signal length is a power of two
 */
void transformSignal(vector<hipDoubleComplex>& signal){
    int diff = isPowerOfTwo(signal.size()) ? 0 : findNextPowerOfTwo(signal.size()) - signal.size();
    if(diff == 0) {
        return;
    }else{
        for(int i = 0; i < diff; i++){
            signal.push_back(make_hipDoubleComplex(0,0));	
        }
    }
}

// https://en.wikipedia.org/wiki/Cooley%E2%80%93Tukey_FFT_algorithm
// cd is a complex double
/*
 * Bit reversal algorithm for the iterative version of fft.
 * this is needed because we are doing a bottom up implementation isntead of 
 * top down like we did with the recursive fft
 *  
 */
__device__ unsigned int bit_reversal(unsigned int i, int log2n){
    int rev = 0;
    for (int j = 0; j < log2n; j++) {
        rev <<= 1;
        rev |= (i & 1);
        i >>= 1;
    }
    return rev;
}

/**
 * Discrete Fourier Transform
 *
 * CUDA version of dft.
 *
 * - Requires less overhead, but is slower than fft
 */
__global__ void dft_kernal(const hipDoubleComplex* a, hipDoubleComplex* A, unsigned int N) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < N) {
        A[k] = make_hipDoubleComplex(0.0, 0.0);
        for (int t = 0; t < N; t++) {
            double angle = 2 * M_PI * t * k / N;
            // Perform a Euler conversion to do cuda operations
            hipDoubleComplex c = make_hipDoubleComplex(cos(angle), sin(angle));
            A[k] = hipCadd(A[k], hipCmul(a[t], c));
        }
    }
}

/*
 * Perform a Fast Fourier Transform (FFT) on audio data.
 * 
 * 2-radix fft transform
 *
 * My brain hurt.
 *
 * Output will be a vector.
 */
__global__ void iterative_fft_kernel(const hipDoubleComplex* a, hipDoubleComplex* A, int log2n){
    
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    A[i] = a[bit_reversal(i, log2n)];

    __syncthreads();

    for (int s = 1; s <= log2n; ++s) {
        int m = 1 << s;
        int m2 = m >> 1;

        //Indexing for the two inner loops from the original function
        int j = threadIdx.x % m2;
        int k = threadIdx.x / m2 * m;

        float tempr;
        float tempi;

        //Puts the sin into tempi and cos into tempr
        //Idea came from the relationship between sin and cos and exp functions
        //We modifed the initial algorithm with Eulers
        //https://en.wikipedia.org/wiki/Euler%27s_formula
        sincos(j * (3.1415926536/m2), &tempi, &tempr);
        
        //We then put make w a complex double to match the original algorithm 
        hipDoubleComplex w = make_hipDoubleComplex(tempr, tempi);

        hipDoubleComplex t = hipCmul(w, A[k+j + m2]);
        hipDoubleComplex u = A[k+j];
        A[k+j] = hipCadd(u,t);
        A[k+j + m2] = hipCsub(u,t);
        __syncthreads();
    }
}

/*
 * Handles the cuda operations and calls iterative_fft_kernel
 */
int fft_cuda(const hipDoubleComplex* a, hipDoubleComplex* A, int log2n, unsigned int N){    
    // Allocate memory on the cuda device
    hipDoubleComplex* a0;
    hipDoubleComplex* A0;
    hipMalloc((void **)&a0, sizeof(hipDoubleComplex) * N);
    hipMalloc((void **)&A0, sizeof(hipDoubleComplex) * N);
    
    hipMemcpy(a0, a, sizeof(hipDoubleComplex) * N, hipMemcpyHostToDevice);

    //Just putting this here for now dont feel like making testsing stuff atm
    //Just gets the max amount of threads possible given the cuda device
    hipDeviceProp_t properties;    hipGetDeviceProperties(&properties, 0);
    
    int block_size;
    int min_block_size;
    hipOccupancyMaxPotentialBlockSize(&min_block_size, &block_size, iterative_fft_kernel, 0, N);
    int block_count = (N + block_size -1)/block_size;

    START_TIMER(fft)
    iterative_fft_kernel<<<block_count, block_size>>>(a0, A0, log2n);
    hipDeviceSynchronize();
    STOP_TIMER(fft)

    hipMemcpy(A, A0, sizeof(hipDoubleComplex)* N, hipMemcpyDeviceToHost);
    
    printf("Parallel CUDA - Transform Type: FFT - Time: %lfs\n", GET_TIMER(fft));
    hipFree(a0);
    hipFree(A0);

    return 0;
}

/*
 * Handle dft CUDA operations
 */
int dft_cuda(const hipDoubleComplex* a, hipDoubleComplex* A, unsigned int N){
    // Allocate memory on the cuda device
    hipDoubleComplex* a0;
    hipDoubleComplex* A0;
    hipMalloc((void **)&a0, sizeof(hipDoubleComplex) * N);
    hipMalloc((void **)&A0, sizeof(hipDoubleComplex) * N);
    
    hipMemcpy(a0, a, sizeof(hipDoubleComplex) * N, hipMemcpyHostToDevice);

    //Just putting this here for now dont feel like making testsing stuff atm
    //Just gets the max amount of threads possible given the cuda device
    hipDeviceProp_t properties;    hipGetDeviceProperties(&properties, 0);
    
    int block_size;
    int min_block_size;
    hipOccupancyMaxPotentialBlockSize(&min_block_size, &block_size, dft_kernal, 0, N);
    int block_count = (N + block_size -1)/block_size;

    START_TIMER(dft)
    dft_kernal<<<block_count, block_size>>>(a0, A0, N);
    hipDeviceSynchronize();
    STOP_TIMER(dft)

    hipMemcpy(A, A0, sizeof(hipDoubleComplex)* N, hipMemcpyDeviceToHost);
    
    printf("Parallel CUDA - Transform Type: DFT - Time: %lfs\n", GET_TIMER(dft));
    hipFree(a0);
    hipFree(A0);

    return 0;
}

/*
 * Python helper function to plot fourier transform
 */
void plotOutputData(){
    system("python3 ./python-stuffs/plotter.py");
}

/*
 * Write data to a CSV file
 *
 * File will be parsed in visualizer
 * 
 * Return: The number of complex numbers ie number of lines, will make wrapping with the visalizer easier
 */
int writeDataToCSVFile(const vector<complex<double>>& out, const string fileName = "coords.csv"){

    ofstream outFile(fileName);
    outFile << "x,y" << "\n";
    int count = 0;
//    double max_real = numeric_limits<double>::min();
//    double max_imag = numeric_limits<double>::min();
    for(complex<double> i : out){
        count++;
        outFile << i.real() << "," << i.imag() << "\n";

//        if (i.real() > max_real){
//            max_real = i.real();
//        } 
//        if (i.imag() > max_imag){
//            max_imag = i.imag();
//        }
    }

    outFile.close();

    // plotOutputData();
    // normalizeCSVFile(out, max_real, max_imag, fileName);
    return count;
}

/*
 * Write data to a CSV file
 *
 * File will be parsed in visualizer
 * 
 * Return: The number of complex numbers ie number of lines, will make wrapping with the visalizer easier
 */
int writeDataToCSVFile(hipDoubleComplex* out, int outsize,const string fileName = "coords.csv"){

    ofstream outFile(fileName);
    outFile << "x,y" << "\n";
    int count = 0;
   // double max_real = numeric_limits<double>::min();
   // double max_imag = numeric_limits<double>::min();
    for(int i = 0; i < outsize; i++){
        count++;
        outFile << out[i].x << "," << out[i].y << "\n";
    }
  //      if (out[i].x > max_real){
  //         max_real = out[i].x;
  //      } 
  //      if (out[i].y > max_imag){
  //          max_imag = out[i].y;
  //      }
  //  }

    outFile.close();

    // plotOutputData();
    // normalizeCSVFile(out, max_real, max_imag, fileName);
    return count;
}

int main(int argc,const char** argv){
    cin.tie(0);

    // Read input file
    if (argc != 3) {
   	printf("Usage: %s <input-wav-file-name> <output-csv-file-name>\n args = %d", argv[0],argc); 
	exit(EXIT_FAILURE);
    }

    std::string file_name = argv[1];
    std::string csv_name  = argv[2];
    std::string dft_csv_name = csv_name.substr(0, csv_name.size() - 4) + "_dft.csv";

    vector<hipDoubleComplex> output = getDataFromWav2(file_name);    
    transformSignal(output); // Ensure that output size is a power of 2

    // Convert the vector to array, yes I know not optimal
    hipDoubleComplex fft_in[output.size()];
    hipDoubleComplex dft_in[output.size() * 2];
    copy(output.begin(),output.end(), fft_in);
    copy(output.begin(),output.end(), dft_in);

    hipDoubleComplex* fft_out;
    fft_out = (hipDoubleComplex*)calloc(output.size(),sizeof(hipDoubleComplex));
    hipDoubleComplex* dft_out;
    dft_out = (hipDoubleComplex*)calloc(output.size(),sizeof(hipDoubleComplex));
    
    int log2n = log2(output.size());

    fft_cuda(fft_in, fft_out, log2n, output.size());

    dft_cuda(dft_in, dft_out, output.size());

    
    // Serial for now - TODO: Add OMP def
    writeDataToCSVFile(fft_out, output.size(), csv_name);
    writeDataToCSVFile(dft_out, output.size(), dft_csv_name);
    free(fft_out);
    free(dft_out);

    return EXIT_SUCCESS;
}