#include "hip/hip_runtime.h"
#include "dependencies.h"


// Default starting values for cRe and cIm
float const REAL = -0.7;
float const IMAGINARY = 0.27015;
float cRe = REAL;
float cIm = IMAGINARY;

// Dimensions of the window are set in the display function
// This is done soe we can use glutGet
int const height = 1024;
int const width = 1024;
int max_iterations = 150;

double zoom = 1;
double mx = 0;
double my = 0;

int animation = 0;
double t = 0.0;

int step_To_Seek = 0;

//Background color;
float br = 0.0, bg = 0.0, bb = 0.0;
// GLubyte* pixels;
float pixels[1024][1024][3];

// float ***pixels;

void display();
void animate();
void julia(double zoom, double mX, double mY);
void key_listener(unsigned char key, int x, int y);
void arrow_listener(int key, int x, int y);
double random_interval() {return (double)rand()/(double)RAND_MAX;}
void print_stats();
double* csv_to_array(char* file); 

int main( int argc, char** argv )
{
    // csv_to_array("cat.csv");

    glutInit( &argc, argv );
    glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGB);
    glutInitWindowSize( width, height);
    glutInitWindowPosition(950,0);
    glutCreateWindow( "Julia" );

    glutDisplayFunc(animate);
    glutIdleFunc(animate);
    glutKeyboardFunc(key_listener);
    glutSpecialFunc(arrow_listener);
    glutMainLoop();
    // free(pixels);
    return 0;
}

__global__
void julia(double zoom, double mX, double mY)
{
    
	double zx, zy, ox, oy;
    int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x *  gridDim.x;

    // glBegin( GL_POINTS ); // start drawing in single pixel mode, 
    // KILLS performance but should be easily parallizable with cuda I hope
    // If failure to parallelize with cuda with draw with triangles and textures instead

    // algorithm to draw the julia set. 
    // basic pseduo code can be found at https://en.wikipedia.org/wiki/Julia_set
    // Note this algorithm is modified
	// for (int y = 0; y < height; y++){ // Draws one frame.
	// 	for (int x = 0; x < width; x++){
	zx = 1.5 * (x - width / 2) / (0.5 * zoom * width) + mX;
	zy = (y - height / 2) / (0.5 * zoom * height) + mY;
	int iteration = 0;
	for (iteration; iteration < max_iterations; iteration++){
		ox = zx;
		oy = zy;
		zx = (ox * ox - oy * oy) + cRe;
		// zy = (2 * ox * oy + cIm);
        zy = (ox * oy + ox * oy) + cIm;
		if((zx * zx + zy * zy) > 4) break;
	}	
		if(iteration == max_iterations ){// Set color to draw julia
        // glColor3f( oy, zx - zy, oy-ox); 
        // glColor3f(ox + oy, oy, abs(ox-oy)); // Set the color of everything not part of the julia set
        // glVertex2i( x, y );
        pixels[y][x][0] = oy;
        pixels[y][x][1] = zx-zy;
        pixels[y][x][2] = oy-ox;
	}
    else { // Set color to draw pixels not apart of julia
        // glColor3f(br,bg,bb);
        pixels[y][x][0] = 0.0;
        pixels[y][x][1] = 0.0;
        pixels[y][x][2] = 0.0;
        // glColor3f(abs(oy -ox), ox, ox + oy);
        // glColor3f(abs(ox-oy) + 0.1,0.0,abs(ox-oy) + .01);
        // glVertex2i( x, y );
    }
	// 	}
	// }
    // glEnd();
}

void animate(){
  
    if (animation == 1){ 
        
        double *buf;
        buf = csv_to_array("cat.csv");

        double old_time = t;
        t = clock();
        double delta_time = (t - old_time);

        // cRe = (cRe + buf[0]/1000000) + 0.005 * sin(delta_time/zoom);
        // cIm = (cIm + buf[1]/10000000) + 0.005 * cos(delta_time/zoom); 

        cRe = (cRe ) + 0.0001 * sin(delta_time + 10);
        cIm = (cIm ) + 0.0001 * cos(delta_time + 10); 
        // printf("test %f  %f\n", cRe, cIm);
        // br = .01 * remainder(buf[0],step_To_Seek) / 255;
        // bg = .01 * remainder(buf[1],step_To_Seek) / 255;
        // bb = .01 * remainder(cRe, cIm)/255;

    }
    display();
}

void display()
{
    // height = glutGet( GLUT_WINDOW_HEIGHT );
    // width = glutGet( GLUT_WINDOW_WIDTH );
    dim3 grids(1024/16, 1024,16);
    dim3 threads(16,16);

    glClearColor(0.0, 0.0, 0.0, 0.0);
    glClear( GL_COLOR_BUFFER_BIT );
    glDrawPixels(width, height, GL_RGB, GL_FLOAT, pixels);

    // julia(zoom, mx, my);
    julia<<<grids, threads>>>(zoom, mx,my);
    glutSwapBuffers();
}

void key_listener(unsigned char key, int x, int y){
    switch(key){
        case 'q':
            // free(pixels);
            exit(0);
            break;
        case 'w':
            max_iterations += 10;
            break;
        case 's':
            if (max_iterations > 0){
                max_iterations -= 10;
            }
            break;
        //Adjust real
        case 'r':
            cRe += .1;
            break;
        case 'f':
            cRe -= .1;
            break;
        //Adjust Imaginary
        case 'i':
            cIm += .01;
            break;
        case 'k': 
            cIm -= .01;
            break;
        case 32: // Spacebar / Toggle animation
            if (animation == 0){
                animation = 1;
            } else {
                animation = 0;
            }
            break;
        case '=': // Zoom Camera In 
            zoom += .1;
            break;
        case '-': // Zoom Camera Out
            zoom -= .1;
            break;
        default:
            // printf("Key id_%d is not a valid input\n", key);
            printf("Valid keys:\n\
            q = exit\n\
            w = increment iterations by 10\n\
            s = decrement iterations by 10\n\
            r = increment real by .1\n\
            s = decrement real by .1 \n\
            i = increment imaginary by .01\n\
            k = decrement imaginary by .01\n\
            space = enable/disable animation\n\
            = = zoom in\n - = zoom out\n\
            ArrowKeys to pan camera in a direction\n\
            ");
            return;
    }

    // printf("Iterations %d", max_iterations);
    print_stats();

}

void arrow_listener(int key, int x, int y){
    switch(key){
        case GLUT_KEY_LEFT:
            mx -= .1;
            break;
        case GLUT_KEY_RIGHT:
            mx += .1;
            break;
        case GLUT_KEY_UP:
            my += .1;
            break;
        case GLUT_KEY_DOWN:
            my -= .1;
            break;
    }

    print_stats();
}

void print_stats(){
    printf("Iterations %d   real = %f   imaginary = %f   animation = %d\n", max_iterations, cRe, cIm, animation);
    printf("    Camera: pos = %f, %f   zoom = %f\n", mx,my,zoom);
}

double* csv_to_array(char *file){
    static double ret[2];
    FILE *fp;
    fp = fopen(file, "r");
    if (!fp){
        puts("File Not Found");
        exit(EXIT_FAILURE);
    }

    fseek(fp, step_To_Seek, SEEK_SET);
    char *token;
    char buffer[200];
    
    fgets(buffer, 200, fp);
    token = strtok(buffer, ",");
    int i = 0;
    int len = 0;
    while (token != NULL){
        // printf("Token = %s", token);
        len += strlen(token);
        ret[i] = strtod(token, &token);
        token = strtok(NULL, ",");
        i += 1;
    }

    step_To_Seek = (len + step_To_Seek) % 4126;

    return ret;
}