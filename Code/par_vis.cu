#include "hip/hip_runtime.h"
#include "GPU_Animation_Bitmap.h"
#include "dependencies.h"

#define block 1024

double cRe = -0.7;
double cIm = 0.27015;
double t = 0.0;
int max_iterations = 150;
double zoom = 1;
double mx = 0;
double my = 0;
int animation = 0;

int Step_To_Seek = 0;

GPUAnimBitmap *bitmap_Ptr;
unsigned int frames = 0;

void key_listener(unsigned char key, int x, int y);
void arrow_listener(int key, int x, int y);
void print_stats(float render_time);
double* csv_to_array(char* file); 

__global__
void julia(uchar4 *pixels, int max_iterations,
double cRe, double cIm, double mX, double mY, double zoom) {	
    double zx, zy, ox, oy;
	int height = block;
	int width = block;

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x *  gridDim.x;
    // printf("offset = %d\n", offset);
    // printf("x(%d) = threadIdx.y(%d) + blockIdx.x(%d) * blockDim.x(%d)\n", x, threadIdx.x, blockIdx, blockDim.x);
    // printf("y(%d) = threadIdx.x(%d) + blockIdx.y(%d) * blockDim.y(%d)\n", x, threadIdx.x, blockIdx, blockDim.x);
    // printf("block dim = %d   \n gridDim.x = %d  offset = %d\n", blockDim.x, gridDim.x, offset);

	zx = 1.5 * (x - width / 2) / (0.5 * zoom * width) - mX;
	zy = (y - height / 2) / (0.5 * zoom * height) + mY;

	int iteration = 0;
	while (iteration < max_iterations){
		ox = zx;
		oy = zy;
		zx = (ox * ox - oy * oy) + cRe;
        zy = (ox * oy + ox * oy) + cIm;
		if((zx * zx + zy * zy) > 4) {
			pixels[offset].x = 0.0;
			pixels[offset].y = 0.0;
			pixels[offset].z = 0.0;
            pixels[offset].w = 255;

            // pixels[offset + 0] = 0;
            // pixels[offset + 1] = 0;
            // pixels[offset + 2] = 0;
            // pixels[offset + 3] = 255;

			return;
		}

		iteration++;
	}	

	if (iteration == max_iterations){
		// pixels[offset].x = (unsigned char) 255 * oy;
		// pixels[offset].y = (unsigned char) 255 * (zx - zy);
		// pixels[offset].z = (unsigned char) 255 * (oy - ox);
        // pixels[offset].w = 255;

        pixels[offset].x = (unsigned char) 255 * 0;
		pixels[offset].y = (unsigned char) 255 * (ox*ox);
		pixels[offset].z = (unsigned char) 255 * oy;
        pixels[offset].w = 255;

	}
}

void generateFrame(uchar4 *ptr) {
	dim3 grids(block/16, block/16);
	dim3 threads(16, 16);
    if (animation == 1){ 
        
        double *buf;
        buf = csv_to_array("cat.csv");

        double old_time = t;
        t = clock();
        double delta_time = (t - old_time);

        cRe = (cRe + buf[0]/10000000)  + 0.005 * sin(delta_time/zoom);
        cIm = (cIm + buf[1]/100000000) + 0.005 * cos(delta_time/zoom); 

        // br = .01 * remainder(buf[0],step_To_Seek) / 255;
        // bg = .01 * remainder(buf[1],step_To_Seek) / 255;
        // bb = .01 * remainder(cRe, cIm)/255;
        // print_stats();
    }

    START_TIMER(julia);
	julia<<<grids, threads>>>(ptr, max_iterations, cRe, cIm, mx, my, zoom);
    hipDeviceSynchronize();
    STOP_TIMER(julia);
    print_stats(GET_TIMER(julia));
    // julia(ptr, max_iterations, cRe, cIm, mx, my, zoom);
    frames++;
}

int main(void) {
	GPUAnimBitmap bitmap(block, block, NULL);
    bitmap_Ptr = &bitmap;
	
	bitmap.anim_and_exit((void (*)(uchar4*, void*))generateFrame, NULL);
	glutKeyboardFunc(key_listener);
	glutSpecialFunc(arrow_listener);
	glutMainLoop();
}

void key_listener(unsigned char key, int x, int y){
    switch(key){
        case 'q':
            // Free our buffers and cuda mem
            bitmap_Ptr->free_resources();
            exit(0);
            break;
        case 'w':
            max_iterations += 10;
            break;
        case 's':
            if (max_iterations > 0){
                max_iterations -= 10;
            }
            break;
        //Adjust real
        case 'r':
            cRe += .1;
            break;
        case 'f':
            cRe -= .1;
            break;
        //Adjust Imaginary
        case 'i':
            cIm += .01;
            break;
        case 'k': 
            cIm -= .01;
            break;
        case 32: // Spacebar / Toggle animation
            if (animation == 0){
                animation = 1;
            } else {
                animation = 0;
            }
            break;
        case '=': // Zoom Camera In 
            zoom += .1;
            break;
        case '-': // Zoom Camera Out
            zoom -= .1;
            break;
        default:
            // printf("Key id_%d is not a valid input\n", key);
            printf("Valid keys:\n\
            q = exit\n\
            w = increment iterations by 10\n\
            s = decrement iterations by 10\n\
            r = increment real by .1\n\
            s = decrement real by .1 \n\
            i = increment imaginary by .01\n\
            k = decrement imaginary by .01\n\
            space = enable/disable animation\n\
            = = zoom in\n - = zoom out\n\
            ArrowKeys to pan camera in a direction\n\
            ");
            return;
    }
        // print_stats();

    }

void arrow_listener(int key, int x, int y){
    switch(key){
        case GLUT_KEY_LEFT:
            mx -= .1;
            break;
        case GLUT_KEY_RIGHT:
            mx += .1;
            break;
        case GLUT_KEY_UP:
            my += .1;
            break;
        case GLUT_KEY_DOWN:
            my -= .1;
            break;
    }
    // print_stats();
}

void print_stats(float render_time){
    printf("Time To Render = %f:\n", render_time);
    printf("Iterations %d   real = %f   imaginary = %f   animation = %d\n", max_iterations, cRe, cIm, animation);
    printf("    Camera: pos = %f, %f   zoom = %f\n", mx,my,zoom);
}

double* csv_to_array(char *file){
    static double ret[2];
    FILE *fp;
    fp = fopen(file, "r");
    if (!fp){
        puts("File Not Found");
        exit(EXIT_FAILURE);
    }

    fseek(fp, Step_To_Seek, SEEK_SET);
    char *token;
    char buffer[200];
    
    fgets(buffer, 200, fp);
    token = strtok(buffer, ",");
    int i = 0;
    int len = 0;
    while (token != NULL){
        // printf("Token = %s", token);
        len += strlen(token);
        ret[i] = strtod(token, &token);
        token = strtok(NULL, ",");
        i += 1;
    }

    Step_To_Seek = (len + Step_To_Seek) % 4126;

    return ret;
}
